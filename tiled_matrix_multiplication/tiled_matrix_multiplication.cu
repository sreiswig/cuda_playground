#include "hip/hip_runtime.h"
#define TILE_WIDTH 16
__global__ void matrixMulKernel(float * M, float * N, float * P, int Width) {
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  // Identify the row and column of the P element to work on
  int Row = by * TILE_WIDTH + ty;
  int Column = bx * TILE_WIDTH + tx;

  // Loop over the M and N tiles required to compute P element
  float Pvalue = 0;
  for (int ph = 0; ph < Width/TILE_WIDTH; ++ph) {

    // Collaborative loading of M and N tiles into shared memory
    Mds[ty][tx] = M[Row*Width + ph*TILE_WIDTH + tx];
    Nds[ty][tx] = N[(ph*TILE_WIDTH + ty) * Width + Col];
    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; ++k) {
      Pvalue += Mds[ty][k] * Nds[k][tx];
    }
    __syncthreads();
  }
  P[Row*Width + Col] = Pvalue;
} 
