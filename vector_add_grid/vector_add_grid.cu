#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Handling arbitrary vector size
  if (tid < n) {
    out[tid] = a[tid] + b[tid];
  }
}

int main(){
  
  int nDevices;
  hipGetDeviceCount(&nDevices);
  
  printf("Number of devices: %d\n", nDevices);
  
  for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device Number: %d\n", i);
      printf("  Device name: %s\n", prop.name);
      printf("  Memory Clock Rate (MHz): %d\n",
           prop.memoryClockRate/1024);
      printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
      printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
      printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
      printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
      printf("  minor-major: %d-%d\n", prop.minor, prop.major);
      printf("  Warp-size: %d\n", prop.warpSize);
      printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
      printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
    }

    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    printf("a[0] = %f\n", a[0]);
    printf("b[0] = %f\n", b[0]);

    // Executing kernel 
    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);
    vector_add<<<grid_size, block_size>>>(d_out, d_a, d_b, N);

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    printf("out[0] = %f\n", out[0]);

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
