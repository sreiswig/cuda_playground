#include <iostream>
#include <bits/stdc++.h>
#include <fstream>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>


__global__ void vector_diff(float *out, float *a, float *b, int n) {
  for(int i = 0; i < n; i++) {
    out[i] = a[i] - b[i];
  }
}

int main() {
  std::string line;
  std::vector<float> list1;
  std::vector<float> list2;
  std::ifstream input ("input.txt");

  if (input.is_open()) {
    while (std::getline(input, line, ' ')) {
      list1.push_back(std::stof(line));
      std::getline(input, line);
      list2.push_back(std::stof(line));
    }
    input.close();
  }

  int elements = list1.size();
  float *a = list1.data();
  float *b = list2.data();
  float *out = (float*)malloc(sizeof(float) * elements);

  float *d_a, *d_b, *d_out;

  hipMalloc((void**)&d_a, sizeof(float) * elements);
  hipMalloc((void**)&d_b, sizeof(float) * elements);
  hipMalloc((void**)&d_out, sizeof(float) * elements);

  hipMemcpy(d_a, a, sizeof(float) * elements, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float) * elements, hipMemcpyHostToDevice);

  vector_diff<<<1,1>>>(d_out, d_a, d_b, elements);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("CUDA Error: %s\n", hipGetErrorString(err));
  }

  hipMemcpy(out, d_out, sizeof(float) * elements, hipMemcpyDeviceToHost);
  printf("out[0] = %f\n", out[0]);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);

  free(out);
}
